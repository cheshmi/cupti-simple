#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cupti.h>

void checkCUPTIError(CUptiResult err, const char *msg) {
    if (err != CUPTI_SUCCESS) {
        const char *errStr;
        cuptiGetResultString(err, &errStr);
        fprintf(stderr, "%s: %s\n", msg, errStr);
        exit(-1);
    }
}




int main() {
    CUptiResult cuptiStatus;

    // Subscribe to CUPTI events
    //cupti_subscription_id subscriptionId;
 //   cuptiSubscribe(&subscriptionId, CUPTI_ACTIVITY_KIND_KERNEL, callback, &activityHandle);
  
    // Initialize CUPTI
//    cuptiStatus = cuptiSubscribe(); //cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)callbackFunction, NULL);
//    checkCUPTIError(cuptiStatus, "Failed to subscribe to CUPTI");

    return 0;
}



